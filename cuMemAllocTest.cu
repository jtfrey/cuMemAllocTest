#include "hip/hip_runtime.h"
#include <stdio.h>
#include <errno.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>

#ifdef HAVE_MPI
#include <mpi.h>

#define PRINTF_FORMAT_HEADER    "[%04d:%04d] "
#define PRINTF_ARG_HEADER       splitRank, splitSize,
#else
#define PRINTF_FORMAT_HEADER
#define PRINTF_ARG_HEADER
#endif

#ifndef GPU_CONTEXT_FLAGS
#define GPU_CONTEXT_FLAGS (0)
#endif

unsigned int gpuContextFlags = GPU_CONTEXT_FLAGS;

int
main(
    int               argc,
    const char*       argv[]
)
{
    int               argi, indexWidth, topRank, topSize, splitRank, splitSize;
    int               gpuCount = 0, gpuIndex;
    hipDevice_t          gpuDevice;
    hipCtx_t         gpuContext;
    size_t            gpuMemSize;
    hipError_t          rc;
    const char        *rcStr;

#ifdef HAVE_MPI
    MPI_Comm          shmcomm;
#endif

    if ( argc < 2 ) {
        fprintf(stderr, "usage:\n\n    %s <mem_size_bytes> {<mem_size_bytes> ..}\n\n", argv[0]);
        return 1;
    }
    indexWidth = 1 + floor(log10(argc - 1));

#ifdef HAVE_MPI
    MPI_Init_thread(&argc, (char***)&argv, MPI_THREAD_SERIALIZED, &argi);
    MPI_Comm_size(MPI_COMM_WORLD, &topSize); splitSize = topSize;
    MPI_Comm_rank(MPI_COMM_WORLD, &topRank); splitRank = topRank;
    fprintf(stderr, PRINTF_FORMAT_HEADER "%0*d: MPI top-level initialization, %d of %d\n", PRINTF_ARG_HEADER indexWidth, 0, topRank, topSize);
    
    MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, 0, MPI_INFO_NULL, &shmcomm);
    MPI_Comm_size(shmcomm, &splitSize);
    MPI_Comm_rank(shmcomm, &splitRank);
    fprintf(stderr, PRINTF_FORMAT_HEADER "%0*d: MPI shared comm initialization, %d of %d\n", PRINTF_ARG_HEADER indexWidth, 0, splitRank, splitSize);
#endif

    rc = hipInit(0);
    hipDrvGetErrorString(rc, &rcStr); 
    fprintf(stderr, PRINTF_FORMAT_HEADER "%0*d: hipInit(0) = %s (%d)\n", PRINTF_ARG_HEADER indexWidth, 0, rcStr, rc);
    if ( rc != hipSuccess ) return 1;

    rc = hipGetDeviceCount(&gpuCount);
    hipDrvGetErrorString(rc, &rcStr);
    fprintf(stderr, PRINTF_FORMAT_HEADER "%0*d: hipGetDeviceCount(%p) = %s (%d) :: gpuCount = %d\n", PRINTF_ARG_HEADER indexWidth, 0, &gpuCount, rcStr, rc, gpuCount);
    if ( (rc != hipSuccess) || (gpuCount < 1) ) return 1;
    
    gpuIndex = splitRank % gpuCount;
    fprintf(stderr, PRINTF_FORMAT_HEADER "%0*d: will use device at index %d\n", PRINTF_ARG_HEADER indexWidth, 0, gpuIndex);
    
    rc = hipDeviceGet(&gpuDevice, gpuIndex);
    hipDrvGetErrorString(rc, &rcStr);
    fprintf(stderr, PRINTF_FORMAT_HEADER "%0*d: cuGetDevice(%p, %d) = %s (%d)\n", PRINTF_ARG_HEADER indexWidth, 0, &gpuDevice, gpuIndex, rcStr, rc);
    if ( rc != hipSuccess ) return 1;

    rc = hipDeviceTotalMem(&gpuMemSize, gpuDevice);
    hipDrvGetErrorString(rc, &rcStr);
    fprintf(stderr, PRINTF_FORMAT_HEADER "%0*d: hipDeviceTotalMem(%p, %p) = %s (%d) :: gpuMemSize = %lld\n", PRINTF_ARG_HEADER indexWidth, 0, &gpuMemSize, gpuDevice, rcStr, rc, (long long)gpuMemSize);
    if ( rc != hipSuccess ) return 1;

    rc = hipCtxCreate(&gpuContext, gpuContextFlags, gpuDevice);
    hipDrvGetErrorString(rc, &rcStr);
    fprintf(stderr, PRINTF_FORMAT_HEADER "%0*d: hipCtxCreate(%p, %x, %p) = %s (%d)\n", PRINTF_ARG_HEADER indexWidth, 0, &gpuContext, gpuContextFlags, gpuDevice, rcStr, rc);
    if ( rc != hipSuccess ) return 1;

    argi = 1;
    while ( argi < argc ) {
        hipDeviceptr_t   gpuMem;
        long          reqMemSize;
        char          *endptr;

        if ( strchr(argv[argi], '%') ) {
            double    percentage = strtod(argv[argi], &endptr);

            if ( (percentage <= 0.0) || (endptr == argv[argi]) ) {
                fprintf(stderr, "ERROR:  invalid memory percentage: %s\n", argv[argi]);
                return errno;
            }
            reqMemSize = floor(gpuMemSize * 0.01 * percentage);
        } else if ( strcmp(argv[argi], "all") == 0 ) {
            reqMemSize = gpuMemSize;
        } else {
            reqMemSize = strtol(argv[argi], &endptr, 0);
            if ( (reqMemSize == 0) && (endptr == argv[argi]) ) {
                fprintf(stderr, "ERROR:  invalid memory size: %s\n", argv[argi]);
                return errno;
            }
            if ( reqMemSize == 0 ) reqMemSize = gpuMemSize;
            else if ( reqMemSize < 0 ) reqMemSize = floor((1.0 - 0.125) * gpuMemSize);
        }
        fprintf(stderr, PRINTF_FORMAT_HEADER "%0*d: selected memory size %ld\n", PRINTF_ARG_HEADER indexWidth, argi, reqMemSize);
        rc = hipMalloc(&gpuMem, (size_t)reqMemSize);
        hipDrvGetErrorString(rc, &rcStr);
        fprintf(stderr, PRINTF_FORMAT_HEADER "%0*d: hipMalloc(%p, %ld) = %s (%d)\n", PRINTF_ARG_HEADER indexWidth, argi, &gpuMem, reqMemSize, rcStr, rc);
        if ( rc == hipSuccess) {
            rc = hipFree(gpuMem);
            hipDrvGetErrorString(rc, &rcStr);
            fprintf(stderr, PRINTF_FORMAT_HEADER "%0*d: hipFree(%p) = %s (%d)\n", PRINTF_ARG_HEADER indexWidth, argi, gpuMem, rcStr, rc);
        }
        argi++;
    }

    rc = hipCtxDestroy(gpuContext);
    hipDrvGetErrorString(rc, &rcStr);
    fprintf(stderr, PRINTF_FORMAT_HEADER "%0*d: hipCtxDestroy(%p) = %s (%d)\n", PRINTF_ARG_HEADER indexWidth, 0, gpuContext, rcStr, rc);
    if ( rc != hipSuccess ) return 1;

#ifdef HAVE_MPI
    MPI_Finalize();
#endif

    return 0;
}

